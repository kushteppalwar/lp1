#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

__global__ void matmul(int* M, int* N, int* P, int* width)
{
  int row = blockIdx.x;
  int col = threadIdx.x;
  P[row*(*width)+col]=0;    //set the output of current element to 0
  for(int i=0; i<*width; i++)
  {
    P[row*(*width)+col] += M[row*(*width)+i]*N[i*(*width)+col]; //I've converted the general A[row][col] to A[row*width+col]
  }                                                             //because of the row major format
}
//d_xyz in my code means xyz is on the device
int main()
{
  int width = 4;  //width of n*n matrix
  int* d_width;
  hipMalloc(&d_width, sizeof(int));
  //copy width
  hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
  
  //define input matrices
  int M[width][width] = {{5,7,9,10},
                        {2,3,3,8},  
                        {8,10,2,3},
                        {3,3,4,8}
                        };

  int N[width][width] = {{3,10,12,18},
                        {12,1,4,9},
                        {9,10,12,2},
                        {3,12,4,10}};
  
  //declare output matrix on host side
  int P[width][width];

  int *d_M, *d_N, *d_P;
  hipMalloc(&d_M, sizeof(int)*width*width);
  hipMalloc(&d_N, sizeof(int)*width*width);
  hipMalloc(&d_P, sizeof(int)*width*width);

  //copy matrices to GPU
  hipMemcpy(d_M, M, sizeof(int)*width*width, hipMemcpyHostToDevice);
  hipMemcpy(d_N, N, sizeof(int)*width*width, hipMemcpyHostToDevice);
  hipMemcpy(d_P, P, sizeof(int)*width*width, hipMemcpyHostToDevice);
  
  matmul<<<width, width>>>(d_M, d_N, d_P, d_width);     
  hipMemcpy(P, d_P, sizeof(int)*width*width, hipMemcpyDeviceToHost);
  
  cout<<"The output is:\\n";
  for(int i=0; i<width; i++)
  {
    for(int j=0; j<width; j++)
    {
      cout<<P[i][j]<<" ";
    }
    cout<<"\\n";
  }
  hipFree(d_M);
  hipFree(d_N);
  hipFree(d_P);
  return 0;
}

