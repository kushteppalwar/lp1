
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__
void sum(int *input){
	int tid = threadIdx.x;
	int step =1;
	int number_of_threads = blockDim.x;
	while(number_of_threads>0){
		if(tid<number_of_threads){
			int fst = tid * step * 2;
			int snd = fst + step;
      printf("%d\\n",input[fst]+input[snd]);
			input[fst]+=input[snd];
      
		}
		step *=2;
		number_of_threads/=2;
	}
	
}

int main(){
	int count = 8;
	int size = count * sizeof(int);
	
	int h[] = {10,20,30,40,50,60,70,80};
	int *d_h;
	hipMalloc(&d_h,size);
	hipMemcpy(d_h,h,size,hipMemcpyHostToDevice);
	
	sum<<<1,count/2>>>(d_h);
	int result;	hipMemcpy(&result,d_h,sizeof(int),hipMemcpyDeviceToHost);
	cout<<result;
	hipFree(d_h);
}
